#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "MUtils.h"

cudaTimer::cudaTimer() {
        hipEventCreate(&_start);
        hipEventCreate(&_stop);
}

cudaTimer::~cudaTimer() {
        hipEventDestroy(_start);
        hipEventDestroy(_stop);
}

void cudaTimer::start() {
        hipEventRecord(_start, 0);
}

void cudaTimer::stop() {
        hipEventRecord(_stop, 0);
        hipEventSynchronize(_stop);
}

float cudaTimer::getElapsedTimeMillis() {
        float time;
        hipEventElapsedTime(&time, _start, _stop);
        return time;
}

void exitWithMsg(const char *msg, int exitCode) {
        printf("ERROR\n");
        printf("%s\n\n", msg);
        exit(exitCode);
}

int _ConvertSMVer2Cores(int major, int minor) {
    typedef struct {
        int SM;
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
            return nGpuArchCoresPerSM[index].Cores;

        index++;
    }

    return -1;
}

CUDAcard findBestDevice() {
        int numOfDevices, bestDeviceNumber;

        hipDeviceProp_t bestDeviceProps;

        safeAPIcall(hipGetDeviceCount(&numOfDevices), __LINE__);

        int maxCores = -1;

        for (int i = 0; i < numOfDevices; ++i) {
                hipDeviceProp_t currentDeviceProps;
                safeAPIcall(hipGetDeviceProperties(&currentDeviceProps, i), __LINE__);

                int deviceCores = _ConvertSMVer2Cores(currentDeviceProps.major,
                                currentDeviceProps.minor) * currentDeviceProps.multiProcessorCount;

                if (maxCores < deviceCores) {
                        maxCores = deviceCores;
                        bestDeviceNumber = i;
                        bestDeviceProps = currentDeviceProps;
                }
        }

        if(maxCores < 0 || numOfDevices < 1)
                exitWithMsg("No CUDA capable card detected.", -2);

        CUDAcard gpu;
        gpu.cardNumber = bestDeviceNumber;
        gpu.major = bestDeviceProps.major;
        gpu.minor = bestDeviceProps.minor;
        gpu.cardsInSystem = numOfDevices;
        gpu.maxThreadsPerBlock = bestDeviceProps.maxThreadsDim[0];
        gpu.SMs = bestDeviceProps.multiProcessorCount;
        gpu.cudaCores = maxCores;
        gpu.globalMem = bestDeviceProps.totalGlobalMem;
        strcpy(gpu.name, bestDeviceProps.name);

        return gpu;
}

void safeAPIcall(hipError_t err, int line) {
        if(err != hipSuccess) {
                printf("Error in line %d\n", line);
                exitWithMsg(hipGetErrorString(err), -2);
        }
}

void printCardInfo(CUDAcard gpu) {
        printf("\t>Name: %s\n", gpu.name);
        printf("\t>CUDA Capability: %d.%d\n", gpu.major, gpu.minor);
        printf("\t>Global memory: %.0f MBytes\n", (float) gpu.globalMem / 1048576.0f);
        printf("\t>Multiprocessors: %d\n", gpu.SMs);
        printf("\t>CUDA cores: %d\n", gpu.cudaCores);
}

