#include <stdio.h>
#include <stdlib.h>
#include "dna.h"
#include "device.h"
#include "MUtils.h"
#include <iostream>
using namespace std;
//#include "device.cu"
//define the chunk sizes that each threadblock will work on
#define DATAXSIZE 671
#define DATAYSIZE 671
#define DATAZSIZE 671

#define BLKXSIZE 4
#define BLKYSIZE 4
#define BLKZSIZE 4

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)


int main(int argc, char *argv[])
{
    typedef int nRarray[DATAYSIZE][DATAXSIZE];
    DNA sequence1,sequence2,sequence3;
    int scores[4]={5,2,-5,-4},*d_scores,*d_allStates;
    int allStates[27]={111,112,113,121,122,123,131,132,133,211,212,213,221,222,223,231,232,233,311,312,313,321,322,323,331,332,333};
    sequence1.read_file(argv[1]);
    sequence2.read_file(argv[2]);
    sequence3.read_file(argv[3]);
    char *d_sequence1,*d_sequence2,*d_sequence3;


    printf("\n Sequence 1: %s \t Length: %d",sequence1.seq_string,sequence1.seq_length);
    printf("\n Sequence 2: %s \t Length: %d",sequence2.seq_string,sequence2.seq_length);
    printf("\n Sequence 3: %s \t Length: %d",sequence3.seq_string,sequence3.seq_length);
    const int nx = sequence1.seq_length;
    const int ny = sequence2.seq_length;
    const int nz = sequence3.seq_length;
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize(((nx+BLKXSIZE-1)/BLKXSIZE), ((ny+BLKYSIZE-1)/BLKYSIZE), ((nz+BLKZSIZE-1)/BLKZSIZE));

   unsigned size=nx*ny*nz;
    nRarray *c; // storage for result stored on host
    nRarray *d_c;  // storage for result computed on device
// allocate storage for data set
   if ((c = (nRarray *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
// allocate GPU device buffers
   hipMalloc((void **) &d_sequence1, (nx)*sizeof(char));	
   cudaCheckErrors("Failed to allocate device buffer");
   hipMemcpy(d_sequence1, &sequence1.seq_string, (nx)*sizeof(char), hipMemcpyHostToDevice);
   cudaCheckErrors("Failed to copy device buffer");
	
   hipMalloc((void **) &d_sequence2, (ny)*sizeof(char));
   cudaCheckErrors("Failed to allocate device buffer");
   hipMemcpy(d_sequence2, &sequence2.seq_string, (ny)*sizeof(char), hipMemcpyHostToDevice);
   cudaCheckErrors("Failed to copy device buffer");

   hipMalloc((void **) &d_sequence3, (nz)*sizeof(char));
   cudaCheckErrors("Failed to allocate device buffer");
   hipMemcpy(d_sequence3, &sequence3.seq_string, (nz)*sizeof(char), hipMemcpyHostToDevice);
   cudaCheckErrors("Failed to copy device buffer");

    hipMalloc((void **) &d_c, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_scores, 4*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_allStates, 27*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMemcpy(d_scores, &scores, (4*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy device buffer");
    hipMemcpy(d_allStates, &allStates, (27*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy device buffer");
// compute result
    cudaTimer kernelTimer;
    kernelTimer.start();
    parallel_score<<<gridSize, blockSize>>>(d_sequence1,d_sequence2,d_sequence3,nx,ny,nz,d_c,size,d_scores,d_allStates);
    cudaCheckErrors("Kernel launch failure");
    hipDeviceSynchronize();
    //parallel_score<<<gridSize, blockSize>>>(d_c,size,d_scores,d_allStates);
    kernelTimer.stop();
    double gcups = (sequence1.seq_length * sequence2.seq_length * sequence3.seq_length)/(1e9 * kernelTimer.getElapsedTimeMillis());
    cout<<"\n Kernel Time:"<<kernelTimer.getElapsedTimeMillis()/1000<<"\n GCUPS:"<<gcups<<endl;

    hipMemcpy(c, d_c, ((nx*ny*nz)*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    free(c);
    hipFree(d_c);
    hipFree(d_scores);
    hipFree(d_allStates);
    cudaCheckErrors("hipFree fail");
    return 0;
}
