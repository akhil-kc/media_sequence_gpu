#include <stdio.h>
#include <stdlib.h>
#include "dna.h"
#include "device.h"
#include "MUtils.h"
#include "stateTransition.h"
#include <iostream>
#include <getopt.h>
using namespace std;
//#include "device.cu"
//define the chunk sizes that each threadblock will work on
#define DATAXSIZE 671
#define DATAYSIZE 671
#define DATAZSIZE 671

#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

static struct option options[] = {
    {"sequence1", required_argument, 0, 'a'},
    {"sequence2", required_argument, 0, 'b'},
    {"sequence3", required_argument, 0, 'c'},
    {"match", required_argument, 0, 'm'},
    {"insertion", required_argument, 0, 'i'},
    {"deletion", required_argument, 0, 'd'},
    {"mismatch", required_argument, 0, 'n'},
    {"help", no_argument, 0, 'h'},
    {0, 0, 0, 0}
};

static void help() {
    printf(
    "Usage: ./median -a <sequence 1> -b <sequence 2> -c <sequence 3> [Arguments ...]\n"
    "\n"
    "Arguments:\n"
    "\t -a, --sequence1 <file> (required)\n"
    "\t\t Input Sequence 1 FASTA format\n"
    "\t -b, --sequence2 <file> (required)\n"
    "\t\t Input Sequence 2 FASTA format\n"
    "\t -c, --sequence3 <file> (required)\n"
    "\t\t Input Sequence 3 FASTA format\n"
    "\t -m, --match <int> (default : 5)\n"
    "\t\t Match score, should be a positive integer \n"
    "\t -n, --mismatch <int> (default : -10)\n"
    "\t\t Mismatch score, should be a negative integer \n"
    "\t -d, --deletion <int> (default : -5)\n"
    "\t\t Deletion score, should be a negative integer \n"
    "\t -i, --insertion <int> (default : -4)\n"
    "\t\t Insertion score, should be a negative integer \n"
    "\t -h, -help\n"
    "\t\t prints out the help\n");
}

int main(int argc, char *argv[])
{
    CUDAcard gpu;
    typedef int nRarray[DATAYSIZE][DATAXSIZE];
    char *seq1_path=NULL;
    char *seq2_path=NULL;
    char *seq3_path=NULL;
    DNA sequence1,sequence2,sequence3;
    int scores[4]={5,-10,-5,-4},*d_scores,*d_allStates;
    //1 represents Match, 2 represents Insert, 3 represents Delete
    int allStates[27]={111,222,113,223,221,131,232,212,133,211,213,231,233,311,122,311,313,121,123,321,323,331,112,132,312,332,333};

    if (argc<3){
	help();
	return 0;
     }

     while (1) {
        char argument = getopt_long(argc, argv, "a:b:c:m:n:d:i:h", options, NULL);
        if (argument == -1) {
            break;
        }
        switch (argument) {
        case 'a': seq1_path= optarg;
            	break;
        case 'b': seq2_path= optarg;
	    	break;
	case 'c': seq3_path= optarg;
	    	break;
	case 'm': scores[0]=  atoi(optarg);
		break;
	case 'n': scores[1]=  atoi(optarg);
		break;
	case 'i': scores[2]=  atoi(optarg);
		break;
	case 'd': scores[3]=  atoi(optarg);
		break;
	case 'h':
	default: help();
		return -1;
    	}
    }
    sequence1.read_file(seq1_path);
    sequence2.read_file(seq2_path);
    sequence3.read_file(seq3_path);
    char *d_sequence1,*d_sequence2,*d_sequence3;


    printf("\n ---------------------------------------------------------------------------------------------"); 
    printf("\n Sequence 1: %s \t Length: %d",sequence1.seq_name,sequence1.seq_length);
    printf("\n Sequence 2: %s \t Length: %d",sequence2.seq_name,sequence2.seq_length);
    printf("\n Sequence 3: %s \t Length: %d",sequence3.seq_name,sequence3.seq_length);
    printf("\n ---------------------------------------------------------------------------------------------"); 
    printf("\n GPU Info \n");
    gpu=findBestDevice();
    printCardInfo(gpu);
    printf("\n ---------------------------------------------------------------------------------------------"); 

    const int nx = sequence1.seq_length;
    const int ny = sequence2.seq_length;
    const int nz = sequence3.seq_length;
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize(((nx+BLKXSIZE-1)/BLKXSIZE), ((ny+BLKYSIZE-1)/BLKYSIZE), ((nz+BLKZSIZE-1)/BLKZSIZE));
    short **stateTrans= calculateTransition();
    printf("\n Scoring Values \n ________________ \n Match: %d \n Mismatch: %d \n Insertion: %d \n Deleteion: %d",scores[0],scores[1],scores[3],scores[2]);
    printf("\n ---------------------------------------------------------------------------------------------"); 

   unsigned size=nx*ny*nz;
    nRarray *c; // storage for result stored on host
    nRarray *d_c;  // storage for result computed on device
    nRarray *d_state;  // storage for result computed on device
    nRarray *state;  // storage for result computed on device
// allocate storage for data set
   if ((c = (nRarray *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
   if ((state = (nRarray *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
// allocate GPU device buffers
   hipMalloc((void **) &d_sequence1, (nx)*sizeof(char));	
   cudaCheckErrors("Failed to allocate device buffer");
   hipMemcpy(d_sequence1, &sequence1.seq_string, (nx)*sizeof(char), hipMemcpyHostToDevice);
   cudaCheckErrors("Failed to copy device buffer");
	
   hipMalloc((void **) &d_sequence2, (ny)*sizeof(char));
   cudaCheckErrors("Failed to allocate device buffer");
   hipMemcpy(d_sequence2, &sequence2.seq_string, (ny)*sizeof(char), hipMemcpyHostToDevice);
   cudaCheckErrors("Failed to copy device buffer");

   hipMalloc((void **) &d_sequence3, (nz)*sizeof(char));
   cudaCheckErrors("Failed to allocate device buffer");
   hipMemcpy(d_sequence3, &sequence3.seq_string, (nz)*sizeof(char), hipMemcpyHostToDevice);
   cudaCheckErrors("Failed to copy device buffer");

    hipMalloc((void **) &d_c, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_state, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_scores, 4*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_allStates, 27*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMemcpy(d_scores, &scores, (4*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy device buffer");
    hipMemcpy(d_allStates, &allStates, (27*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy device buffer");
// compute result
    cudaTimer kernelTimer;
    kernelTimer.start();
    parallel_score<<<gridSize, blockSize>>>(d_sequence1,d_sequence2,d_sequence3,nx,ny,nz,d_c,d_state,d_scores,d_allStates);
    cudaCheckErrors("Kernel launch failure");
    hipDeviceSynchronize();
    kernelTimer.stop();
    double gcups = (sequence1.seq_length * sequence2.seq_length * sequence3.seq_length)/(kernelTimer.getElapsedTimeMillis()/1000);
    cout<<"\n Kernel Time:"<<kernelTimer.getElapsedTimeMillis()/1000<<"\n CUPS:"<<gcups<<endl;

    hipMemcpy(c, d_c, ((nx*ny*nz)*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");
   cout<<"\n Last Cell Score: "<<c[nx-1][ny-1][nz-1]<<endl; 
    free(c);
    hipFree(d_c);
    hipFree(d_scores);
    hipFree(d_allStates);
    cudaCheckErrors("hipFree fail");
    return 0;
}
