#include "hip/hip_runtime.h"
#include<stdio.h>
#include "device.h"
#include "dna.h"
#define DATAXSIZE 671
#define DATAYSIZE 671
#define DATAZSIZE 671

__device__ int getScore(char a,char b,char c,int scores[4],int allStates[27])
{
	//1 Represents Match, 2 represents Insert , 3 represents Delete
	int score,maxScore=-999;
	int match=scores[0];
	int mismatch=scores[1];
	int insert=scores[2];
	int del=scores[3];
	 for(int i=0;i<27;i++)
        {
		if (allStates[i]<200)
		{
			switch(allStates[i]){
				case 111: score=((a==b)?((a==c)?(match+match+match):(match+match+mismatch)):((a==c)?(match+mismatch+match):(mismatch+mismatch+mismatch)));
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 112: score=((a==b)?(match+match+insert):(mismatch+mismatch+insert));
					 maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 113: score=((a==b)?(match+match+del):(mismatch+mismatch+del));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 121: score=((a==c)?(match+match+insert):(mismatch+mismatch+insert));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 122: score=(match+insert+insert);
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 123: score=(match+insert+del);
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 131: score=((a==c)?(match+del+match):(mismatch+del+mismatch));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 132: score=(match+insert+del);
					   maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 133: score=(match+del+del);
				          maxScore=(score>maxScore)?score:maxScore;
					  break;
		
			}
		}
		else if (allStates[i]<300)
		{
			switch(allStates[i]){
			case 211: score=((b==c)?(insert+match+match):(insert+mismatch+mismatch));
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 212: score=(insert+match+insert);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 213: score=(insert+match+del);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 221: score=(insert+insert+match);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 222: score=(insert+insert+insert);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 223: score=(insert+insert+del);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 231: score=(insert+del+match);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 232: score=(insert+del+insert);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 233: score=(insert+del+del);
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			}
		}
		else{
			switch(allStates[i]){
			case 311: score=((b==c)?(del+match+match):(del+mismatch+mismatch));
				  maxScore=(score>maxScore)?score:maxScore;
				  break;
			case 312: score=(del+match+insert);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 313: score=(del+match+del);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 321: score=(del+insert+match);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 322: score=(del+insert+insert);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 323: score=(del+insert+del);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 331: score=(del+del+match);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 332: score=(del+del+insert);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
                        case 333: score=(del+del+del);
                                  maxScore=(score>maxScore)?score:maxScore;
                                  break;
			}
		}

	}
	return maxScore;
}

__global__ void parallel_score(char *sequence1,char * sequence2,char* sequence3,int xLen,int yLen,int zLen,int a[][DATAYSIZE][DATAXSIZE],int state[][DATAYSIZE][DATAXSIZE],int scores[4],int allStates[27])
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    if ((idx < (xLen)) && (idy < (yLen)) && (idz < (zLen)) ){
	a[idx][idy][idz]=getScore(sequence1[idx],sequence2[idy],sequence3[idz],scores,allStates);
      }
}

