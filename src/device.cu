#include "hip/hip_runtime.h"
#include<stdio.h>
#include "device.h"
#include "dna.h"
#define DATAXSIZE 671
#define DATAYSIZE 671
#define DATAZSIZE 671

__device__ int getScore(char a,char b,char c,int scores[4],int allStates[27])
{
	//1 Represents Match, 2 represents Insert , 3 represents Delete
	int score,maxScore=-999;
	int match=scores[0];
	int mismatch=scores[1];
	int insert=scores[2];
	int del=scores[3];
	 for(int i=0;i<27;i++)
        {
		if (allStates[i]<200)
		{
			switch(allStates[i]){
				case 111: score=((a==b)?((a==c)?(match+match+match):(match+match+mismatch)):((a==c)?(match+mismatch+match):(mismatch+mismatch+mismatch)));
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 112: score=((a==b)?(match+match+insert):(mismatch+mismatch+insert));
					 maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 113: score=((a==b)?(match+match+del):(mismatch+mismatch+del));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 121: score=((a==c)?(match+match+insert):(mismatch+mismatch+insert));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 122: score=(match+insert+insert);
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 123: score=(match+insert+del);
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 131: score=((a==c)?(match+del+match):(mismatch+del+mismatch));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 132: score=(match+insert+del);
					   maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 133: score=(match+del+del);
						maxScore=(score>maxScore)?score:maxScore;
					  break;
		
			}
		}
	}
	return maxScore;
}

__global__ void parallel_score(char *sequence1,char * sequence2,char* sequence3,int xLen,int yLen,int zLen,int a[][DATAYSIZE][DATAXSIZE],unsigned size,int scores[4],int allStates[27])
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    if ((idx < (xLen)) && (idy < (yLen)) && (idz < (zLen)) ){
	a[idx][idy][idz]=getScore(sequence1[idx],sequence2[idy],sequence3[idz],scores,allStates);
      }
}

