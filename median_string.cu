#include <stdio.h>
#include <stdlib.h>
#include "dna.h"
#include "Stateinfo.h"
#include "MUtils.h"
#include <iostream>
using namespace std;
//#include "device.cu"
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)


int main(int argc, char *argv[])
{
    DNA *sequences=new DNA[3];
    int scores[4]={5,2,-5,-4},*d_scores,*d_allStates;
    int allStates[27]={111,112,113,121,122,123,131,132,133,211,212,213,221,222,223,231,232,233,311,312,313,321,322,323,331,332,333};
    sequences[0].read_file(argv[1]);
    sequences[1].read_file(argv[2]);
    sequences[2].read_file(argv[3]);
    DNA *d_sequences;
    const size_t sz = size_t(3) * sizeof(DNA);
    hipMalloc((void**)&d_sequences,sz);
    hipMemcpy(d_sequences, &sequences, sz, hipMemcpyHostToDevice);
    printf("\n Sequence 1: %s \t Length: %d",sequences[0].seq_string,sequences[0].seq_length);
    printf("\n Sequence 2: %s \t Length: %d",sequences[1].seq_string,sequences[1].seq_length);
    printf("\n Sequence 3: %s \t Length: %d",sequences[2].seq_string,sequences[2].seq_length);
    const int nx = sequences[0].seq_length;
    const int ny = sequences[1].seq_length;
    const int nz = sequences[2].seq_length;
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize(((nx+BLKXSIZE-1)/BLKXSIZE), ((ny+BLKYSIZE-1)/BLKYSIZE), ((nz+BLKZSIZE-1)/BLKZSIZE));

   unsigned size=nx*ny*nz;
// pointers for data set storage via malloc
    int *c; // storage for result stored on host
    int *d_c;  // storage for result computed on device
// allocate storage for data set
   if ((c = (int *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
// allocate GPU device buffers
    hipMalloc((void **) &d_c, (nx*ny*nz)*sizeof(int));
    hipMalloc((void **) &d_scores, 4*sizeof(int));
    hipMalloc((void **) &d_allStates, 27*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMemcpy(d_scores, &scores, (4*sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_allStates, &allStates, (27*sizeof(int)), hipMemcpyHostToDevice);
// compute result
	cudaTimer kernelTimer;
    kernelTimer.start();

    parallel_score<<<gridSize,blockSize>>>(d_sequences,d_c,size,d_scores,d_allStates);
    cudaCheckErrors("Kernel launch failure");
	 kernelTimer.stop();
	 double gcups = (sequences[0].seq_length * sequences[1].seq_length * sequences[2].seq_length)/(1e6 * kernelTimer.getElapsedTimeMillis());
	cout<<"\n Kernel Time:"<<kernelTimer.getElapsedTimeMillis()/1000<<"\n GCUPS:"<<gcups<<endl;
// copy output data back to host
	cout<<"After GPU Call";

    hipMemcpy(c, d_c, ((nx*ny*nz)*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");
	cout<<endl<<c[100]<<endl<<c[600]<<endl<<c[6000];
    printf("Results check!\n");
    free(c);
    hipFree(d_c);
    cudaCheckErrors("hipFree fail");
    return 0;
}
