#include "hip/hip_runtime.h"
#include "Stateinfo.h"
#include "dna.h"

__device__ int getScore(char a,char b,char c,int scores[4],int allStates[27])
{
	//1 Represents Match, 2 represents Insert , 3 represents Delete
	int score,maxScore=-999;
	int match=scores[0];
	int mismatch=scores[1];
	int insert=scores[2];
	int del=scores[3];
	 for(int i=0;i<27;i++)
        {
		if (allStates[i]<200)
		{
			switch(allStates[i]){
				case 111: score=((a==b)?((a==c)?(match+match+match):(match+match+mismatch)):((a==c)?(match+mismatch+match):(mismatch+mismatch+mismatch)));
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 112: score=((a==b)?(match+match+insert):(mismatch+mismatch+insert));
					 maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 113: score=((a==b)?(match+match+del):(mismatch+mismatch+del));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 121: score=((a==c)?(match+match+insert):(mismatch+mismatch+insert));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 122: score=(match+insert+insert);
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 123: score=(match+insert+del);
					  maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 131: score=((a==c)?(match+del+match):(mismatch+del+mismatch));
					  maxScore=(score>maxScore)?score:maxScore;
                                          break;
				case 132: score=(match+insert+del);
					   maxScore=(score>maxScore)?score:maxScore;
					  break;
				case 133: score=(match+del+del);
						maxScore=(score>maxScore)?score:maxScore;
					  break;
		
			}
		}
	}
	return maxScore;
}

__global__ void parallel_score(DNA sequences[3],int *a,unsigned size,int scores[4],int allStates[27])
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    unsigned xLen=sequences[0].seq_length;
    unsigned yLen=sequences[1].seq_length;
    unsigned zLen=sequences[2].seq_length;
    unsigned index= idx + (xLen * (idy + (yLen * idz)));
    if ((idx < (xLen)) && (idy < (yLen)) && (idz < (zLen)) && (index < size)){
	 int z = index / (xLen * yLen);
        index -= (z * xLen * yLen);
        int y = index / xLen;
        int x = index % xLen;
      a[index] = getScore(sequences[0].seq_string[x],sequences[1].seq_string[y],sequences[2].seq_string[z],scores,allStates);
      }
}

